#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "mex.h"

/* Kernel to square elements of the array on the GPU */

__constant__ float cWsum;
__constant__ float cWmean;
__constant__ float cWvar;

#define BLOCK_SIZE 	512			

__global__ void reduce5 ( float * inData, float * outData, int n )
{
	__shared__ float data [BLOCK_SIZE];
	int	  tid = threadIdx.x;
	int	  i   = blockIdx.x * blockDim.x + threadIdx.x;
	
    if( i < n )
        data [tid] = inData [i];		// load into shared memeory
    else
        data [tid] = 0;
	
	__syncthreads ();
	
	for ( int s = blockDim.x / 2; s > 0; s >>= 1 )
	{
		if ( tid < s )
			data [tid] += data [tid + s];
		__syncthreads ();
	}

    if ( tid == 0 )					// write result of block reduction
		outData [blockIdx.x] = data [0];
}

void reduce_array( const float* d_array, int n, float *d_sum, float *h_sum, int block_size )
{
    int i = 0;
    float *dev_arrs[2] = { NULL, NULL };
    hipMalloc ( (void**)&dev_arrs [0], n * sizeof( float ) );
    hipMalloc ( (void**)&dev_arrs [1], n * sizeof( float ) );
    hipMemcpy( dev_arrs[0], d_array, n * sizeof( float ), hipMemcpyDeviceToDevice );
    
    for ( i = 0; n > 1; n = ceil( (float)n / block_size ) , i ^= 1 )
    {
        dim3 dimBlock ( block_size, 1, 1 );
        dim3 dimGrid  ( ceil( (float)n / block_size ), 1, 1 );
        reduce5<<<dimGrid, dimBlock>>> ( dev_arrs[i], dev_arrs[i^1], n );
    }
    
    if ( h_sum != NULL )
    {
        hipMemcpy( h_sum, dev_arrs[i], sizeof( float ), hipMemcpyDeviceToHost );
    }
    
    if ( d_sum != NULL )
    {
        hipMemcpy( d_sum, dev_arrs[i], sizeof( float ), hipMemcpyDeviceToDevice );
    }
    
    hipFree( dev_arrs[0] );
    hipFree( dev_arrs[1] );
}


__global__ void arr_minus_scalar_squared( float* dst, float* src, float scalar, int N )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i < N )
    {
        dst[ i ] = src[ i ] - scalar;
        dst[ i ] = dst[ i ] * dst[ i ];
    }
}

__global__ void compute_mu_sigma( float* K, float *g_Ws, float *g_Xs, int N, int start_index, float S_start, float S_delta )
{
    __shared__ float data [ BLOCK_SIZE ];
    
    int Nmu = blockIdx.x; 
    int Nsigma = blockIdx.y;
    int Nsigmas = gridDim.y;
    int Nmus = gridDim.x; 
    float delta;
    int s;
    int idx = threadIdx.x;
    int i = idx + start_index;        
    float sigma = S_start + Nsigma * S_delta;
    
   
    if( i < N )
    {
        __syncthreads();
        delta = ( g_Xs[ i ] - g_Xs[ Nmu ] );
        __syncthreads();
        data[ idx ] = exp( -delta*delta/(2*sigma*sigma) ) * g_Ws[ i ];
    }
    else
    {
        data[ idx ] = 0;
    }
    
    __syncthreads ();
    
    for ( s = BLOCK_SIZE / 2; s > 0; s >>= 1 )
    {
        if ( (idx < s) )
           data [idx] += data [idx + s];
        __syncthreads ();
    }
    if ( idx == 0 )      
    {
        if ( start_index == 0 )
            K[ Nmu * Nsigmas + Nsigma ] =  data[0] / ( sqrt(2*M_PI) * sigma );
        else
            K[ Nmu * Nsigmas + Nsigma ] +=  data[0] / ( sqrt(2*M_PI) * sigma );
    }
}

__global__ void compute_max_mu_sigma( float* g_max, float *g_Mus, float *g_Ws, float *g_Xs, float *g_Sigmas, int N, int start_index)
{
    __shared__ float data [ BLOCK_SIZE ];
    float delta;
    int s;
    int idx = threadIdx.x;
    int i = idx + start_index;        
    
    float sigma = g_Sigmas[ blockIdx.x ];
    float mu = g_Mus[ blockIdx.x ];
    float xi, wi, tmp, expmax;
    
   
    if( i < N )
    {
        __syncthreads();
        xi = g_Xs[ i ];
        __syncthreads();
        wi = g_Ws[i];
        delta = ( xi - mu );
        data[ idx ] = log( wi ) - delta*delta/(2*sigma*sigma); //exp( -delta*delta/(2*sigma*sigma) );//* g_Ws[ i ] / ( sqrt(2*M_PI) * sigma ) ;
    }
    else
    {
        data[ idx ] = -10000;
    }
    
    __syncthreads ();
   
    for ( s = BLOCK_SIZE / 2; s > 0; s >>= 1 )
    {
        if ( (idx < s) )
           data [idx] = ( data[idx] > data [idx + s] ) ? data[idx] : data [idx + s];
        __syncthreads ();
    }
    if ( idx == 0 )      
    {
        if ( start_index == 0 )
            g_max[ blockIdx.x ] = exp( data[0] ) / ( sqrt(2*M_PI) * sigma ); 
        else
        {
            tmp = g_max[ blockIdx.x ];
            tmp = exp( tmp ) / ( sqrt(2*M_PI) * sigma );
            expmax = exp( data[ 0 ] )/ ( sqrt(2*M_PI) * sigma );
            g_max[ blockIdx.x ] = ( expmax > tmp ) ? expmax : tmp;
        }
    }
}


__global__ void stat_check( float* K, float* Signif, float* g_Xs, float S_start, float S_delta, float M, float alpha, int Npoints )
{
    int Nmu = blockIdx.x; 
    int Nsigma = threadIdx.x;
    int Nsigmas = blockDim.x;
    int Nmus = gridDim.x; 
    
    float mu = g_Xs[ Nmu ];
    float sigma = S_start + Nsigma * S_delta;
    
    float varg;
    float tmu = (1 - mu/M);
    float gmu = 2 * tmu/M * cWsum;
    float gsigma = 0;
    float k, dsp;
       
    float sign;
    __syncthreads();
    k = K[ Nmu * Nsigmas + Nsigma ];
    
    varg = tmu * ( 1/(sqrt(M_PI)*M*sigma) - (1-mu/M)*4/(M*M) );
    dsp = cWvar*(4*tmu*tmu/(M*M)) + cWmean*cWmean*varg + varg*cWvar; 
    gsigma = sqrt( dsp * Npoints );
    
    sign = -log( sqrt(2*M_PI)*gsigma ) - (k - gmu )*(k - gmu )/(2*gsigma*gsigma);
    __syncthreads();
    Signif[ Nmu * Nsigmas + Nsigma ] = sign * ( k > gmu ) * (sign < alpha);
}


void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
int Npoints, m, i, tmp, j, nnz;
double alpha, kN;
double *BigMat;

float MaxMu;
int Nmus;

float 		M;

float *g_max, *g_Mus, *g_Sigmas;
float *h_max, *h_Mus, *h_Sigmas;

float *devSignif, *cpuSignif;
float *devK, *cpuK;
float *Xs_host;
float *Ws_host;

float *g_Xs, *g_Ws;
float *g_tmp;
double *output, *output_ml;

float sigma_start = 1, dsigma = 1;
int Nsigmas = 10;
int Nblocks;
int isig;
double *output_t;

float tWs, tWm, tWv;

dim3 dimGrid, dimBlock;


if ( nrhs != 8 )
    mexErrMsgTxt("Input should == 8");

m = mxGetN(prhs[0]);
Npoints = mxGetM(prhs[0]);
Nblocks = ceil( (float)Npoints / BLOCK_SIZE );
if ( m != 2 )
    mexErrMsgTxt("Wrong weights");

BigMat = mxGetPr( prhs[0] );
alpha = mxGetScalar(prhs[1]);
kN =    mxGetScalar(prhs[2]);
sigma_start =    mxGetScalar(prhs[3]);
dsigma =         mxGetScalar(prhs[4]);
Nsigmas =         int( mxGetScalar(prhs[5]) );
M =         float( mxGetScalar(prhs[6]) );
MaxMu = float( mxGetScalar( prhs[7] ) );
Nmus = 0;

Xs_host = ( float * ) mxMalloc( sizeof(float) * Npoints );
Ws_host = ( float * ) mxMalloc( sizeof(float) * Npoints );

for ( i = 0; i < Npoints; i++ )
{
    Xs_host[i] = (float)BigMat[ i ];
    Ws_host[i] = (float)BigMat[ Npoints + i ];
    if ( Xs_host[i] <= MaxMu )
        Nmus++;
}


cpuSignif = ( float * ) mxMalloc( sizeof(float) * Nmus * Nsigmas );
cpuK = ( float * ) mxMalloc( sizeof(float) * Nmus * Nsigmas );


hipMalloc( (void **) &g_Xs, sizeof( float ) * Npoints );
hipMalloc( (void **) &g_Ws, sizeof( float ) * Npoints );
hipMalloc( (void **) &g_tmp, sizeof( float ) * Npoints );

hipMemcpy( g_Xs, Xs_host, sizeof( float )*Npoints, hipMemcpyHostToDevice );
hipMemcpy( g_Ws, Ws_host, sizeof( float )*Npoints, hipMemcpyHostToDevice );

hipMalloc( (void **) &devSignif, sizeof(float) * Nsigmas * Nmus );
hipMalloc( (void **) &devK, sizeof(float) * Nsigmas * Nmus );

//calculate W_sum
reduce_array( g_Ws, Npoints, NULL, &tWs, BLOCK_SIZE );
hipMemcpyToSymbol(HIP_SYMBOL( cWsum), &tWs, sizeof( float ), 0, hipMemcpyHostToDevice );
//Now calculate W_mean
tWm = (float)tWs/Npoints;
hipMemcpyToSymbol(HIP_SYMBOL( cWmean), &tWm, sizeof( float ), 0, hipMemcpyHostToDevice );
//deviation
arr_minus_scalar_squared<<<dim3(Nblocks),dim3(BLOCK_SIZE)>>>( g_tmp, g_Ws, tWm, Npoints );
reduce_array( g_tmp, Npoints, NULL, &tWv, BLOCK_SIZE );
tWv = tWv / ( Npoints - 1 );
hipMemcpyToSymbol(HIP_SYMBOL( cWvar), &tWv, sizeof( float ), 0, hipMemcpyHostToDevice );
//mexPrintf("\nS=%f M=%f D=%f log(alpha)=%f\n", tWs, tWm, tWv, log(alpha) );
#ifdef TIME_MEASURE
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    hipEventRecord ( start, 0 );    
#endif
    
for ( i = 0; i < Npoints; i += BLOCK_SIZE )
{
    dimBlock.x = BLOCK_SIZE;
    dimGrid.x = Nmus;
    dimGrid.y = Nsigmas;
    compute_mu_sigma<<<dimGrid,dimBlock>>>( devK, g_Ws, g_Xs, Npoints, i, sigma_start, dsigma );
    hipDeviceSynchronize ();
}
    #ifdef TIME_MEASURE
    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );
    printf("c_mu_sigma: %.2f millseconds\n", gpuTime );
    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
    #endif
    
dimGrid.y=1;
dimGrid.x=Nmus;
dimBlock.x=Nsigmas;
    #ifdef TIME_MEASURE
    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    hipEventRecord ( start, 0 );  
    #endif
stat_check<<<dimGrid,dimBlock>>>( devK, devSignif, g_Xs, sigma_start, dsigma, M, log(alpha), Npoints );
    #ifdef TIME_MEASURE
    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );
    printf("stat_check: %.2f millseconds\n", gpuTime );
    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
    #endif
    
hipDeviceSynchronize ();
hipMemcpy( cpuSignif, devSignif, sizeof(float) * Nmus * Nsigmas, hipMemcpyDeviceToHost);
hipMemcpy( cpuK, devK, sizeof(float) * Nmus * Nsigmas, hipMemcpyDeviceToHost);
hipDeviceSynchronize ();



output = ( double * ) mxMalloc( sizeof( double ) * Nsigmas * Nmus * 4 );

nnz = 0;
for ( i = 0; i < Nmus; i++ )
{
    for ( j = 0; j < Nsigmas; j++ )
    {
        if( cpuSignif[ i*Nsigmas + j ] )
        {
            //check that k > max() > 2
            output[ nnz*4 + 0 ] = (double) Xs_host[ i ];
            output[ nnz*4 + 1 ] = (double) sigma_start + dsigma * j;
            output[ nnz*4 + 2 ] = (double) cpuSignif[ i*Nsigmas + j ];
            output[ nnz*4 + 3 ] = (double) cpuK[ i*Nsigmas + j ];
            nnz++;
        }
    }    
}
//Copy all data for k / max() > 2 check
h_max = ( float * ) mxMalloc( sizeof(float) * nnz );
h_Mus = ( float * ) mxMalloc( sizeof(float) * nnz );
h_Sigmas = ( float * ) mxMalloc( sizeof(float) * nnz );
hipMalloc( (void **) &g_max, sizeof( float ) * nnz );
hipMalloc( (void **) &g_Mus, sizeof( float ) * nnz );
hipMalloc( (void **) &g_Sigmas, sizeof( float ) * nnz );
for ( i = 0; i < nnz; i++ )
{
    h_max[ i ] = 0;
    h_Mus[ i ]    = output[ i*4 + 0 ];
    h_Sigmas[ i ] = output[ i*4 + 1 ];
}
hipMemcpy( g_max, h_max,  sizeof(float)* nnz, hipMemcpyHostToDevice);
hipMemcpy( g_Mus, h_Mus,  sizeof(float)* nnz, hipMemcpyHostToDevice);
hipMemcpy( g_Sigmas, h_Sigmas,  sizeof(float)* nnz, hipMemcpyHostToDevice);

#ifdef TIME_MEASURE
 hipEventCreate ( &start );
 hipEventCreate ( &stop );
 hipEventRecord ( start, 0 );  
#endif
 
for ( i = 0; i < Npoints; i += BLOCK_SIZE )
{
    dimBlock.x = BLOCK_SIZE;
    dimBlock.y = 1;
    dimGrid.x = nnz;
    dimGrid.y = 1;
    hipDeviceSynchronize ();
    compute_max_mu_sigma<<<dimGrid,dimBlock>>>( g_max, g_Mus, g_Ws, g_Xs, g_Sigmas, Npoints, i );
    hipDeviceSynchronize ();
}
 #ifdef TIME_MEASURE
    hipEventRecord ( stop, 0 );
    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );
    printf("max: %.2f millseconds\n", gpuTime );
    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
 #endif
    
hipMemcpy( h_max, g_max,  sizeof(float) * nnz, hipMemcpyDeviceToHost );

output_t = ( double * ) mxMalloc( sizeof( double ) * nnz * 4 );
isig = 0;
 
//   for ( i = 0; i < Nmus; i++ )
//   {
//       for ( j = 0; j < Nsigmas; j++ )
//           mexPrintf("%lf ", cpuSignif[ i*Nsigmas + j ] );
//       mexPrintf("\n" );
//   }

for ( i = 0; i < nnz; i++ )
{
    //if ( cpuK[i] / h_max[i] > kN)
   // mexPrintf("%d %f mu %f si %f k %f \n", i, h_max[i], h_Mus[i], h_Sigmas[i], output[ i*4 + 3 ] );
    if ( (output[ i*4 + 3] / h_max[i]) > kN)
    {
        output_t[ isig*4 + 0] = output[ i*4 + 0 ];
        output_t[ isig*4 + 1] = output[ i*4 + 1 ];
        output_t[ isig*4 + 2] = output[ i*4 + 2 ];
        output_t[ isig*4 + 3] = output[ i*4 + 3 ];
        isig++;
    }
}


      


plhs[0] = mxCreateDoubleMatrix( isig,4,mxREAL);
output_ml = mxGetPr(plhs[0]);
for( i = 0; i < isig; i++ )
{
    output_ml[0*isig + i] = output_t[ i*4 + 0 ];
    output_ml[1*isig + i] = output_t[ i*4 + 1 ];
    output_ml[2*isig + i] = output_t[ i*4 + 2 ];
    output_ml[3*isig + i] = output_t[ i*4 + 3 ];
}


//mxSetPr( mxGetPr(plhs[0]), output_ml );

hipFree( devSignif );
hipFree( devK );
hipFree( g_Ws );
hipFree( g_Xs );
hipFree( g_tmp );
hipFree( g_max );
hipFree( g_Mus );
hipFree( g_Sigmas );
mxFree( h_Sigmas );
mxFree( h_Mus );
mxFree( h_max );
mxFree( cpuK );
mxFree( cpuSignif );
mxFree( output );
mxFree( Xs_host );
mxFree( Ws_host );
mxFree( output_t );
/*
data2 = mxGetPr(plhs[i]);
hipEventRecord ( stop, 0 );
hipEventSynchronize ( stop );
hipEventElapsedTime ( &gpuTime, start, stop );
mexPrintf( "time spent executing by the GPU: %.2f millseconds\n", gpuTime );

for (j = 0; j < m*n; j++)
{
data2[j] = (double) data2f[j];
}


mxFree(data1f);
mxFree(data2f);
hipFree(data1f_gpu);
hipFree(data2f_gpu);*/

}