#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "mex.h"

#define BLOCK_SIZE 256
#define MAX_PAT_LEN 100

__constant__ int cpat_len;
__constant__ int cmus[ MAX_PAT_LEN ];
__constant__ float csigmas[ MAX_PAT_LEN ];
__constant__ int cpat_events[ MAX_PAT_LEN ];

__device__ float loss_function(int misses, float lambda)
{
    int b;
    b = (misses != cpat_len);
    return b * exp( - lambda * (float)misses/cpat_len );
}


__global__ void compute_Lh( float *LH, int Nt, int* EventN, int** EventIndexes,  float lambda, float minlh, float denom  )
{
    __shared__ float data [ BLOCK_SIZE ];
    int idx = threadIdx.x;
    int min_d = Nt;
    int min_d_sign = 1;
    int d = Nt;
    int event_index;
    int eps = BLOCK_SIZE * blockIdx.x + idx;
    int bexpr, i, is_missed;
    float delta, tlogLH;
    int tshift=0, prevrx=-1, shift=0, nev;
    int misses = 0;
    float Lhc = 0;
    int sdelta = 0, first_e;
    int min_ind, patev_n;
        
    if ( eps >= Nt ) return;
    
    //if ( idx < EventN[0] )
    //    data[ idx ] = EventIndexes[0][ idx ];
     
    for ( nev = 0; nev < cpat_len; nev++ )
    {
        min_d = Nt;
        min_d_sign = 1;
        tshift = eps + shift + cmus[ nev ];
        patev_n = cpat_events[ nev ] - 1;
        for( i = 0; i < EventN[ patev_n ]; i++ )
        {
            //d = data[i] - eps;
            __syncthreads();
            event_index = EventIndexes[ patev_n ][i];
            d = abs( event_index - tshift);
            bexpr = ( d < min_d ) && (event_index > prevrx);// && ( d >= 0 );
            min_d =  bexpr * d + ( 1 - bexpr ) * min_d;
            min_d_sign = bexpr * (event_index > tshift ? +1: -1) + 
                         (1 - bexpr) * min_d_sign;
            //min_ind = bexpr * event_index + ( 1 - bexpr ) * min_ind;
        }
        
       // first_e = ( nev==0 ) * min_ind + ( nev != 0 ) * first_e;
        
        prevrx = tshift + min_d * min_d_sign; 
        delta = min_d; 
        tlogLH = -delta*delta/(2*csigmas[nev]*csigmas[nev]);
        
        //if
        is_missed = tlogLH < minlh;
        misses += is_missed;
        shift = ( 1 - is_missed ) * ( shift + cmus[nev] + min_d * min_d_sign ) + 
                ( is_missed     ) * ( tshift - eps );
        prevrx = ( 1 - is_missed ) * prevrx + 
                ( is_missed     ) *  tshift;
        Lhc += (1-is_missed) * tlogLH;
    }
    if( eps < Nt )
        LH[ eps ] = loss_function( misses, lambda) * exp( Lhc )/ denom;// / ow( sqrt(2*M_PI)*csigmas[0] ); 
}   

/* Gateway function */
void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{
int i,j;
float minlh;
int N_event_types;
int *event_N;
int **event_indexes;
int Nt;
int pat_len;
int *pat_events;
float  *sigmas;
int *mus;
float lambda;
double *pr;
mxArray *mArray;

int *devEventN;
int **devEventIndexes;
int **devEventIndexesT;
int *devTmp;
float *devLH;

float denom;

float *hostLH;

if (nrhs != 5)
    mexErrMsgTxt("Input should == 5");

N_event_types = mxGetN(prhs[0]);
event_indexes = (int**) mxMalloc( N_event_types * sizeof(int *) );
event_N = (int*) mxMalloc( N_event_types * sizeof(int) );

hipMalloc( (void **) &devEventN,       sizeof( int ) * N_event_types );
hipMalloc( (void **) &devEventIndexes, sizeof( int * ) * N_event_types );
devEventIndexesT = (int**) mxMalloc( N_event_types * sizeof(int *) );

lambda = mxGetScalar( prhs[3] );
minlh = mxGetScalar( prhs[4] );
Nt = mxGetScalar( prhs[1] );

for ( i = 0; i < N_event_types; i++ )
{
    mArray = mxGetField( prhs[0], i,  "indexes" );
    event_N[i] = mxGetN( mArray );
    event_indexes[i] = ( int * ) mxMalloc( event_N[i] *sizeof( int ) );
    for ( j = 0; j < event_N[i]; j++ ) 
        event_indexes[i][j] = ( int ) *( mxGetPr(mArray) + j )-1 ;
    
    hipMalloc( (void **) &devTmp, sizeof( int ) * event_N[i] );
    hipMemcpy( devTmp, event_indexes[i], event_N[i] * sizeof( int ), hipMemcpyHostToDevice );
    devEventIndexesT[i] = devTmp;
}
hipMemcpy( devEventIndexes, devEventIndexesT, N_event_types * sizeof( int* ), hipMemcpyHostToDevice );
hipMemcpy( devEventN, event_N, N_event_types * sizeof( int ), hipMemcpyHostToDevice );


mArray = mxGetField( prhs[2], 0,  "Events" );
pat_len = mxGetN( mArray );
pat_events = ( int * ) malloc( sizeof( int ) * pat_len );
mus = ( int * ) malloc( sizeof( float ) * pat_len );
sigmas = ( float * ) malloc( sizeof( float ) * pat_len );
pr = mxGetPr( mxGetField( prhs[2], 0,  "Theta" ) );

denom = 1;
for ( i = 0; i < pat_len; i++ )
{
    pat_events[ i ] = *( mxGetPr( mxGetField( prhs[2], 0,  "Events" ) ) + i );
    mus[ i ] = (int) pr[ 0 * pat_len + i ];
    sigmas[ i ] = (float) pr[ 1 * pat_len + i ];
    denom *= sigmas[ i ] * sqrt(2*M_PI);
   // mexPrintf("PL %d %d %f\n", pat_events[i], mus[i], sigmas[i]);
}

// hipMemcpyToSymbol(HIP_SYMBOL( cpat_len), &pat_len, sizeof( int ), 0, hipMemcpyHostToDevice );
// hipMemcpyToSymbol(HIP_SYMBOL( cmus), mus, sizeof( int ) * pat_len, 0, hipMemcpyHostToDevice );
// hipMemcpyToSymbol(HIP_SYMBOL( csigmas), sigmas, sizeof( float ) * pat_len, 0, hipMemcpyHostToDevice );
// hipMemcpyToSymbol(HIP_SYMBOL( cpat_events), pat_events, sizeof( float ) * pat_len, 0, hipMemcpyHostToDevice );

hipMemcpyToSymbol(HIP_SYMBOL( "cpat_len"), &pat_len, sizeof( int ), 0, hipMemcpyHostToDevice );
hipMemcpyToSymbol(HIP_SYMBOL( "cmus"), mus, sizeof( int ) * pat_len, 0, hipMemcpyHostToDevice );
hipMemcpyToSymbol(HIP_SYMBOL( "csigmas"), sigmas, sizeof( float ) * pat_len, 0, hipMemcpyHostToDevice );
hipMemcpyToSymbol(HIP_SYMBOL( "cpat_events"), pat_events, sizeof( int ) * pat_len, 0, hipMemcpyHostToDevice );

hostLH = ( float * ) mxMalloc( sizeof( float ) * Nt );
hipMalloc( (void **) &devLH, sizeof( float ) * Nt );

//mexPrintf("%d %d %d %d\n", BLOCK_SIZE, int(ceil((float)Nt/BLOCK_SIZE)), Nt, pat_len );
//mexPrintf("CU %s\n", hipGetErrorString( hipGetLastError() ) );
dim3 dimGrid( int(ceil((float)Nt/BLOCK_SIZE)) );
dim3 dimBlock( BLOCK_SIZE );
hipDeviceSynchronize ();
compute_Lh<<<dimGrid,dimBlock>>>( devLH, Nt, devEventN, devEventIndexes, lambda, minlh, denom );
hipDeviceSynchronize ();
//mexPrintf("CU %s\n", hipGetErrorString( hipGetLastError() ) );
hipMemcpy( hostLH, devLH, sizeof( float )*Nt, hipMemcpyDeviceToHost);
hipDeviceSynchronize ();

plhs[0] = mxCreateDoubleMatrix(Nt,1,mxREAL);
for ( i = 0 ; i < Nt; i++ )
{
   // mexPrintf("%f ", hostLH[i] );
    *(mxGetPr( plhs[0] )+i) = ( double ) hostLH[i];
}
//mexPrintf("CU %s\n", hipGetErrorString( hipGetLastError() ) );
hipFree( devEventN );
hipFree( devLH );
for ( i = 0; i < N_event_types; i++ )
{
    hipFree( devEventIndexesT[ i ] );
    mxFree( event_indexes[ i ] );
}
hipFree( devEventIndexes );
mxFree( hostLH );
mxFree( event_N );
mxFree( event_indexes );
mxFree( devEventIndexesT );
}